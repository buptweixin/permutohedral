#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file permutohedral.cu
 * \brief
 * \author Junyuan Xie
*/

#include "./permutohedral-inl.h"

namespace mxnet {
namespace op {

namespace permutohedral {

template<int key_size>
__global__ void init(CuHashTable<key_size> table,
                     const int n_elements,
                     const float *pos,
                     const float *scale,
                     Pair *matrix) {
  float elevated[key_size+1];
  int greedy[key_size+1];
  int rank[key_size+1];
  float barycentric[key_size+2];
  short key[key_size];

  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_elements) return;

  float sm = 0;
  for (int i = key_size; i > 0; i--) {
    float cf = pos[(i-1)*n_elements + idx]*scale[i-1];
    elevated[i] = sm - i*cf;
    sm += cf;
  }
  elevated[0] = sm;  
    
  // find the closest zero-colored lattice point

  // greedily search for the closest zero-colored lattice point
  short sum = 0;
  for (int i = 0; i <= key_size; i++) {
    float v = elevated[i]*(1.0f/(key_size+1));
    float up = ceilf(v) * (key_size+1);
    float down = floorf(v) * (key_size+1);
    if (up - elevated[i] < elevated[i] - down) {
      greedy[i] = static_cast<short>(up);
    } else {
      greedy[i] = static_cast<short>(down);
    }
    sum += greedy[i];
  }
  sum /= key_size+1;
  
  // sort differential to find the permutation between this simplex and the canonical one
  for (int i = 0; i <= key_size; i++) {
    rank[i] = 0;
    for (int j = 0; j <= key_size; j++) {
      if (elevated[i] - greedy[i] < elevated[j] - greedy[j] ||
          (elevated[i] - greedy[i] == elevated[j] - greedy[j]
           && i > j)) {
        rank[i]++;
      }
    }
  }
  
  if (sum > 0) { // sum too large, need to bring down the ones with the smallest differential
    for (int i = 0; i <= key_size; i++) {
      if (rank[i] >= key_size + 1 - sum) {
        greedy[i] -= key_size+1;
        rank[i] += sum - (key_size+1);
      } else {
        rank[i] += sum;
      }
    }
  } else if (sum < 0) { // sum too small, need to bring up the ones with largest differential
    for (int i = 0; i <= key_size; i++) {
      if (rank[i] < -sum) {
        greedy[i] += key_size+1;
        rank[i] += (key_size+1) + sum;
      } else {
        rank[i] += sum;
      }
    }
  }

  // turn delta into barycentric coords
  for (int i = 0; i <= key_size+1; i++) {
      barycentric[i] = 0;
  }
  
  for (int i = 0; i <= key_size; i++) {
    float delta = (elevated[i] - greedy[i]) * (1.0f/(key_size+1));
    barycentric[key_size-rank[i]] += delta;
    barycentric[key_size+1-rank[i]] -= delta;
  }
  barycentric[0] += 1.0f + barycentric[key_size+1];

  for (int color = 0; color <= key_size; color++) {
    // Compute the location of the lattice point explicitly (all but
    // the last coordinate - it's redundant because they sum to zero)
    for (int i = 0; i < key_size; i++) {
      key[i] = greedy[i] + color;
      if (rank[i] > key_size-color) key[i] -= (key_size+1);
    }

    Pair r;
    r.index = table.insert(key, idx*(key_size+1)+color);
    r.weight = barycentric[color];
    matrix[idx*(key_size+1) + color] = r;
  }
}

template<int key_size, bool normalize>
__global__ void splat(CuHashTable<key_size> table,
                      const int32_t n_elements,
                      const int32_t val_size,
                      float *data,
                      float *val,
                      Pair *matrix) {
  const int idx = threadIdx.y + blockIdx.y * blockDim.y;
  if (idx >= n_elements) return;
  const int color = threadIdx.x;

  Pair r = matrix[idx*(key_size+1)+color];
  float *dst = val + r.index*val_size;
  if (!normalize) {
    for (int j = 0; j < val_size; j++) {
      atomicAdd(dst+j, data[j*n_elements + idx]*r.weight);
    }
  } else {
    for (int j = 0; j < val_size-1; j++) {
      atomicAdd(dst+j, data[j*n_elements + idx]*r.weight);
    }
    atomicAdd(dst+val_size-1, 1.f*r.weight);
  }
}


template<int key_size>
__global__ static void blur(CuHashTable<key_size> table,
                            const int32_t val_size,
                            const int32_t color,
                            float *val,
                            float *new_val, 
                            Pair *matrix) {
  short key[key_size+1];
  short np[key_size+1];
  short nm[key_size+1];
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= table.n_keys_) return;

  // Check if I'm valid
  if (matrix[idx].index != idx) return;

  // find my key and the keys of my neighbours

  for (int i = 0; i < key_size; i++) {
    key[i] = table.keys_[idx*key_size+i];
    np[i] = key[i]+1;    
    nm[i] = key[i]-1;
  }

  np[color] -= key_size+1;
  nm[color] += key_size+1;

  int offNp = table.find(np);
  int offNm = table.find(nm);

  float *valMe = val + val_size*idx;
  float *valNp = val + val_size*offNp;
  float *valNm = val + val_size*offNm; 
  float *valOut = new_val + val_size*idx;

  for (int i = 0; i < val_size; i++) {
    float o = valMe[i];
    if (offNp >= 0) o += 0.5f*valNp[i];
    if (offNm >= 0) o += 0.5f*valNm[i];
    valOut[i] = o;
  }
}

template<int key_size, bool normalize, bool save>
__global__ void slice(CuHashTable<key_size> table,
                      const int32_t n_elements,
                      const int32_t val_size,
                      float *val,
                      float *out,
                      Pair *matrix,
                      float *norm) {
  const float alpha = 1.0f / (1+powf(2, -key_size-1));
  int32_t index[key_size+1];
  float weight[key_size+1];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;    
  if (idx >= n_elements) return;

  for (int i = 0; i <= key_size; ++i) {
    Pair r = matrix[idx*(key_size+1) + i];
    index[i] = r.index;
    weight[i] = r.weight;
  }

  if (!normalize) {
    for (int j = 0; j < val_size; ++j) {
      float v = 0.0f;
      for (int i = 0; i <= key_size; ++i) {
        v += weight[i]*val[index[i]*val_size + j];
      }
      out[j*n_elements + idx] = v * alpha;
    }
  } else {
    float n = 0.0f;
    for (int i = 0; i <= key_size; ++i) {
      n += weight[i]*val[index[i]*val_size + val_size - 1];
    }
    n = 1.0f/n;
    for (int j = 0; j < val_size-1; ++j) {
      float v = 0.0f;
      for (int i = 0; i <= key_size; ++i) {
        v += weight[i]*val[index[i]*val_size + j];
      }
      out[j*n_elements + idx] = v * n;
    }
    if (save)
      norm[idx] = n;
  }
}

template<int key_size, bool normalize>
__global__ void pos_grad_init(const int32_t n_elements, const int32_t val_size, 
                              float *ograd, float *pos, float *data, float *out, float *norm, float *buf) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_elements) return;
  float *f1 = buf;
  float *f2 = f1 + key_size*val_size*n_elements;
  float *f3 = f2 + val_size*n_elements;
  float *f4 = f3 + key_size*val_size*n_elements;

  float p[key_size];
  for (int i = 0; i < key_size; ++i)
    p[i] = pos[i*n_elements + idx];

  float n;
  if (normalize)
    n = norm[idx];
  float deltan = 0.f;

  for (int j = 0; j < (normalize ? val_size - 1 : val_size); ++j) {
    const int idx24 = j*n_elements + idx;
    const float vj = data[idx24];
    const float deltaj = normalize ? ograd[idx24]*n : ograd[idx24];

    f2[idx24] = vj;
    f4[idx24] = deltaj;

    if (normalize)
      deltan -= out[idx24]*deltaj;

    for (int i = 0; i < key_size; ++i) {
      const int idx13 = (i*val_size + j)*n_elements + idx;
      f1[idx13] = p[i]*vj;
      f3[idx13] = p[i]*deltaj;
    }
  }

  if (normalize) {
    const int idx24 = (val_size-1)*n_elements + idx;
    const float vj = 1.f;

    f2[idx24] = vj;
    f4[idx24] = deltan;

    for (int i = 0; i < key_size; ++i) {
      const int idx13 = (i*val_size + val_size-1)*n_elements + idx;
      f1[idx13] = p[i]*vj;
      f3[idx13] = p[i]*deltan;
    }
  }
}

template<int key_size, bool normalize>
__global__ void pos_grad_reduce(const int32_t n_elements, const int32_t val_size,
                                float *ograd, float *pos, float *data, float *out,
                                float *norm, float *buf, float *pgrad) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_elements) return;
  float *f1 = buf;
  float *f2 = f1 + key_size*val_size*n_elements;
  float *f3 = f2 + val_size*n_elements;
  float *f4 = f3 + key_size*val_size*n_elements;

  float p[key_size];
  float pg[key_size];
  for (int i = 0; i < key_size; ++i) {
    p[i] = pos[i*n_elements + idx];
    pg[i] = 0;
  }

  float n;
  if (normalize)
    n = norm[idx];
  float deltan = 0.f;

  for (int j = 0; j < (normalize ? val_size - 1 : val_size); ++j) {
    const int idx24 = j*n_elements + idx;
    const float vj = data[idx24];
    const float deltaj = normalize ? ograd[idx24]*n : ograd[idx24];

    if (normalize)
      deltan -= out[idx24]*deltaj;

    for (int i = 0; i < key_size; ++i) {
      const int idx13 = (i*val_size + j)*n_elements + idx;
      pg[i] += deltaj*f1[idx13] - deltaj*p[i]*f2[idx24]
               + vj*f3[idx13] - vj*p[i]*f4[idx24];
    }
  }

  if (normalize) {
    const int idx24 = (val_size-1)*n_elements + idx;
    const float vj = 1.f;

    for (int i = 0; i < key_size; ++i) {
      const int idx13 = (i*val_size + val_size-1)*n_elements + idx;
      pg[i] += deltan*f1[idx13] - deltan*p[i]*f2[idx24]
               + vj*f3[idx13] - vj*p[i]*f4[idx24];
    }
  }

  for (int i = 0; i < key_size; ++i) {
    pgrad[i*n_elements + idx] = pg[i];
  }
}

}


template<int key_size>
void CuPermutohedralOp<key_size>::GetTempSpace(const OpContext &ctx, int val_size) {
  using namespace mshadow;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();

  Tensor<gpu, 1, uint8_t> tmp =
    ctx.requested[kTemp].get_space_typed<gpu, 1, uint8_t>(
      Shape1(n_keys_*2*sizeof(int32_t) +
             n_keys_*key_size*sizeof(int16_t) +
             n_keys_*val_size*sizeof(float) +
             n_keys_*val_size*sizeof(float) +
             n_keys_*sizeof(Pair)), s);
  uint8_t *ptr = tmp.dptr_;

  int32_t *entries = (int32_t*)ptr;
  entries_ = Tensor<gpu, 1, int32_t>(entries, Shape1(n_keys_*2), s);
  ptr += n_keys_*2*sizeof(int32_t);

  int16_t *keys = (int16_t*)ptr;
  keys_ = Tensor<gpu, 2, int16_t>(keys, Shape2(key_size, n_keys_), s);
  ptr += n_keys_*key_size*sizeof(int16_t);

  float *vals = (float*)ptr;
  vals_ = Tensor<gpu, 2, float>(vals, Shape2(val_size, n_keys_), s);
  ptr += n_keys_*val_size*sizeof(float);

  float *new_vals = (float*)ptr;
  new_vals_ = Tensor<gpu, 2, float>(new_vals, Shape2(val_size, n_keys_), s);
  ptr += n_keys_*val_size*sizeof(float);

  Pair *matrix = (Pair*)ptr;
  matrix_ = Tensor<gpu, 1, Pair>(matrix, Shape1(n_keys_), s);
  ptr += n_keys_*sizeof(Pair);

  CHECK_EQ(ptr, tmp.dptr_ + tmp.shape_.Size());
}

template<int key_size>
void CuPermutohedralOp<key_size>::Filter(hipStream_t stream, permutohedral::CuHashTable<key_size> table, bool normalize, int val_size,
                                         float *scale, float *data, float *pos, float *out, float *norm) {
  using namespace permutohedral;
  
  vals_ = 0;
  if (normalize) {
    splat<key_size, true><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
      table, n_elements_, val_size, data, vals_.dptr_, matrix_.dptr_);
  } else {
    splat<key_size, false><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
      table, n_elements_, val_size, data, vals_.dptr_, matrix_.dptr_);
  }
  CHECK_EQ(hipGetLastError(), hipSuccess);

  float *pval = vals_.dptr_;
  float *pnew_val = new_vals_.dptr_;
  for (int j = 0; j <= key_size; ++j) {
    blur<key_size><<<dim3((n_keys_-1)/lblock_+1, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
      table, val_size, j, pval, pnew_val, matrix_.dptr_);
    CHECK_EQ(hipGetLastError(), hipSuccess);
    std::swap(pval, pnew_val);
  }

  if (normalize) {
    if (norm == NULL) {
      slice<key_size, true, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size, pval, out, matrix_.dptr_, NULL);
    } else {
      slice<key_size, true, true><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size, pval, out, matrix_.dptr_, norm);
    }
  } else {
    slice<key_size, false, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
      table, n_elements_, val_size, pval, out, matrix_.dptr_, NULL);
  }
  CHECK_EQ(hipGetLastError(), hipSuccess);
}

template<int key_size>
void CuPermutohedralOp<key_size>::Forward(const OpContext &ctx,
                                   const std::vector<TBlob> &in_data,
                                   const std::vector<OpReqType> &req,
                                   const std::vector<TBlob> &out_data,
                                   const std::vector<TBlob> &aux_args)  {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  Tensor<gpu, 1, float> scale = aux_args[kScale].get<gpu, 1, float>(s);

  if (!init_) {
    TShape data_shape = in_data[kData].shape_;
    batch_size_ = data_shape[0];
    data_size_ = data_shape[1];
    if (param_.normalize) {
      val_size_ = data_size_ + 1;
    } else {
      val_size_ = data_size_;
    }
    n_elements_ = data_shape.Size()/batch_size_/data_size_;
    n_keys_ = n_elements_*(key_size+1);
    CHECK_EQ(in_data[kPos].size(1), key_size);
    
    lblock_ = cuda::kBaseThreadNum;
    nblock_ = (n_elements_-1)/lblock_+1;

    float cpu_scale[key_size];
    for (int i = 0; i < key_size; i++) {
      cpu_scale[i] = (key_size+1)*sqrtf((2.0/3.0)/((i+1)*(i+2)));
    }
    CHECK_EQ(hipMemcpyAsync((void*)scale.dptr_, (void*)cpu_scale, key_size*sizeof(float), hipMemcpyHostToDevice, stream), hipSuccess);

    init_ = true;
  }
  

  Shape<3> shape = Shape3(batch_size_, data_size_, n_elements_); 
  Tensor<gpu, 3, float> in = in_data[kData].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> out = out_data[kOut].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = key_size;
  Tensor<gpu, 3, float> pos = in_data[kPos].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = 1;
  Tensor<gpu, 3, float> norm = out_data[kNorm].get_with_shape<gpu, 3, float>(shape, s);


  GetTempSpace(ctx, val_size_);

  CuHashTable<key_size> table(n_keys_, entries_.dptr_, keys_.dptr_);


  for (int i = 0; i < batch_size_; ++i) {
    entries_ = -1;

    init<key_size><<<dim3(nblock_, 1, 1), dim3(lblock_,1,1), 0, stream>>>(
      table, n_elements_, pos.dptr_ + i*key_size*n_elements_, scale.dptr_, matrix_.dptr_);
    CHECK_EQ(hipGetLastError(), hipSuccess);

    Filter(stream, table, param_.normalize, val_size_,
           scale.dptr_,
           in.dptr_+i*data_size_*n_elements_,
           pos.dptr_ + i*key_size*n_elements_,
           out.dptr_ + i*data_size_*n_elements_,
           norm.dptr_ + i*n_elements_);
  }
}

template<int key_size>
void CuPermutohedralOp<key_size>::Backward(const OpContext &ctx,
                                           const std::vector<TBlob> &out_grad,
                                           const std::vector<TBlob> &in_data,
                                           const std::vector<TBlob> &out_data,
                                           const std::vector<OpReqType> &req,
                                           const std::vector<TBlob> &in_grad,
                                           const std::vector<TBlob> &aux_args) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  Tensor<gpu, 1, float> scale = aux_args[kScale].get<gpu, 1, float>(s);

  Shape<3> shape = Shape3(batch_size_, data_size_, n_elements_); 
  Tensor<gpu, 3, float> out = out_data[kOut].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> ograd = out_grad[kOut].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> data = in_data[kData].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> data_grad = in_grad[kData].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = key_size;
  Tensor<gpu, 3, float> pos = in_data[kPos].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> pos_grad = in_grad[kPos].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = 1;
  Tensor<gpu, 3, float> norm = out_data[kNorm].get_with_shape<gpu, 3, float>(shape, s);

  GetTempSpace(ctx, req[kPos] == kNullOp ? val_size_ : std::max(val_size_, 2*(key_size+1)*val_size_));

  CuHashTable<key_size> table(n_keys_, entries_.dptr_, keys_.dptr_);

  for (int i = 0; i < batch_size_; ++i) {
    entries_ = -1;

    init<key_size><<<dim3(nblock_, 1, 1), dim3(lblock_,1,1), 0, stream>>>(
      table, n_elements_, pos.dptr_ + i*key_size*n_elements_, scale.dptr_, matrix_.dptr_);
    CHECK_EQ(hipGetLastError(), hipSuccess);

    if (req[kData] != kNullOp) {
      CHECK(req[kData] != kAddTo);
      Filter(stream, table, param_.normalize, val_size_,
             scale.dptr_,
             ograd.dptr_ + i*data_size_*n_elements_,
             pos.dptr_ + i*key_size*n_elements_,
             data_grad.dptr_ + i*data_size_*n_elements_,
             norm.dptr_ + i*n_elements_);
    }

    if (req[kPos] != kNullOp) {
      CHECK(req[kData] != kAddTo);
      if (param_.normalize) {
        pos_grad_init<key_size, true><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
          n_elements_, val_size_,
          ograd.dptr_ + i*data_size_*n_elements_,
          pos.dptr_ + i*key_size*n_elements_,
          data.dptr_ + i*data_size_*n_elements_,
          out.dptr_ + i*data_size_*n_elements_,
          norm.dptr_ + i*n_elements_,
          new_vals_.dptr_);
      } else {
        pos_grad_init<key_size, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
          n_elements_, val_size_,
          ograd.dptr_ + i*data_size_*n_elements_,
          pos.dptr_ + i*key_size*n_elements_,
          data.dptr_ + i*data_size_*n_elements_,
          out.dptr_ + i*data_size_*n_elements_,
          NULL,
          new_vals_.dptr_);
      }
      CHECK_EQ(hipGetLastError(), hipSuccess);

      Filter(stream, table, false, 2*(key_size+1)*val_size_,
             scale.dptr_,
             new_vals_.dptr_,
             pos.dptr_ + i*key_size*n_elements_,
             key_size%2 ? new_vals_.dptr_ : vals_.dptr_,
             NULL);

      if (param_.normalize) {
        pos_grad_reduce<key_size, true><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
          n_elements_, val_size_,
          ograd.dptr_ + i*data_size_*n_elements_,
          pos.dptr_ + i*key_size*n_elements_,
          data.dptr_ + i*data_size_*n_elements_,
          out.dptr_ + i*data_size_*n_elements_,
          norm.dptr_ + i*n_elements_,
          key_size%2 ? new_vals_.dptr_ : vals_.dptr_,
          pos_grad.dptr_ + i*key_size*n_elements_);
      } else {
        pos_grad_reduce<key_size, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
          n_elements_, val_size_,
          ograd.dptr_ + i*data_size_*n_elements_,
          pos.dptr_ + i*key_size*n_elements_,
          data.dptr_ + i*data_size_*n_elements_,
          out.dptr_ + i*data_size_*n_elements_,
          NULL,
          key_size%2 ? new_vals_.dptr_ : vals_.dptr_,
          pos_grad.dptr_ + i*key_size*n_elements_);
      }
      CHECK_EQ(hipGetLastError(), hipSuccess);
    }
  }
}


template<>
Operator *CreateOp<gpu>(PermutohedralParam param, int key_size) {
  switch (key_size) {
   case 2: return new CuPermutohedralOp<2>(param);
   case 3: return new CuPermutohedralOp<3>(param);
   case 4: return new CuPermutohedralOp<4>(param);
   case 5: return new CuPermutohedralOp<5>(param);
   case 6: return new CuPermutohedralOp<6>(param);
   case 7: return new CuPermutohedralOp<7>(param);
   case 8: return new CuPermutohedralOp<8>(param);
   case 9: return new CuPermutohedralOp<9>(param);
   case 10: return new CuPermutohedralOp<10>(param);
   case 11: return new CuPermutohedralOp<11>(param);
   case 12: return new CuPermutohedralOp<12>(param);
   case 13: return new CuPermutohedralOp<13>(param);
   case 14: return new CuPermutohedralOp<14>(param);
   case 15: return new CuPermutohedralOp<15>(param);
   case 16: return new CuPermutohedralOp<16>(param);
   default:
    LOG(FATAL) << "GPU not supported";
    return NULL;
  }
}

}  // namespace op
}  // namespace mxnet

